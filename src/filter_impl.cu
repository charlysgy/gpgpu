#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <cstdio>
#include <iostream>
#include <thread>

#include "filter_impl.h"
#include "logo.h"

// =============================================================================
// CONSTANTS AND MACROS
// =============================================================================

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)

const uint8_t LOW_THRESHOLD = 3;
const uint8_t HIGH_THRESHOLD = 30;

// =============================================================================
// TYPE DEFINITIONS
// =============================================================================

struct rgb
{
    uint8_t r, g, b;
};

struct PixelState
{
    float bg_L, bg_a, bg_b; // Background LAB values
    float cand_L, cand_a, cand_b; // Candidate LAB values
    int t; // Time counter
    int initialized; // Initialization flag
};

// Queue structure for BFS hysteresis
struct Point
{
    int x, y;
};

// =============================================================================
// GLOBAL DEVICE MEMORY
// =============================================================================

__constant__ uint8_t *logo;

// Static variables for pixel state management
static PixelState *d_states = nullptr;
static size_t image_size = 0;

// Queue for BFS hysteresis (allocated once)
static Point *d_queue = nullptr;
static int *d_queue_size = nullptr;
static int *d_next_queue_size = nullptr;
static Point *d_next_queue = nullptr;

// =============================================================================
// UTILITY FUNCTIONS
// =============================================================================

template <typename T>
void check(T err, const char *const func, const char *const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::fprintf(stderr, "CUDA Runtime Error at: %s: %d\n", file, line);
        std::fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);
        std::exit(EXIT_FAILURE);
    }
}

namespace
{
    void load_logo()
    {
        static auto buffer =
            std::unique_ptr<std::byte, decltype(&hipFree)>{ nullptr,
                                                             &hipFree };

        if (buffer == nullptr)
        {
            hipError_t err;
            std::byte *ptr;

            err = hipMalloc(&ptr, logo_width * logo_height);
            CHECK_CUDA_ERROR(err);

            err = hipMemcpy(ptr, logo_data, logo_width * logo_height,
                             hipMemcpyHostToDevice);
            CHECK_CUDA_ERROR(err);

            err = hipMemcpyToSymbol(HIP_SYMBOL(logo), &ptr, sizeof(ptr));
            CHECK_CUDA_ERROR(err);

            buffer.reset(ptr);
        }
    }
} // anonymous namespace

// =============================================================================
// DEVICE UTILITY FUNCTIONS
// =============================================================================

__device__ float srgb_to_linear(float c)
{
    return (c <= 0.04045f) ? __fdiv_rn(c, 12.92f)
                           : __powf(__fdiv_rn(c + 0.055f, 1.055f), 2.4f);
}

__device__ void rgb_to_lab(uint8_t R, uint8_t G, uint8_t B, float &L, float &a,
                           float &b)
{
    // Convert sRGB to linear RGB
    float r = srgb_to_linear(R / 255.0f);
    float g = srgb_to_linear(G / 255.0f);
    float b_ = srgb_to_linear(B / 255.0f);

    // Linear RGB to XYZ conversion
    float X =
        __fmaf_rn(r, 0.4124564f, __fmaf_rn(g, 0.3575761f, b_ * 0.1804375f));
    float Y =
        __fmaf_rn(r, 0.2126729f, __fmaf_rn(g, 0.7151522f, b_ * 0.0721750f));
    float Z =
        __fmaf_rn(r, 0.0193339f, __fmaf_rn(g, 0.1191920f, b_ * 0.9503041f));

    // XYZ to LAB conversion
    float Xn = 0.95047f, Yn = 1.0f, Zn = 1.08883f;
    float inv_Xn = __frcp_rn(Xn);
    float inv_Yn = __frcp_rn(Yn);
    float inv_Zn = __frcp_rn(Zn);

    float fx = (__fmul_rn(X, Xn) > 0.008856f)
        ? cbrtf(__fmul_rn(X, inv_Xn))
        : __fmaf_rn(7.787f, __fmul_rn(X, inv_Xn), 16.0f / 116.0f);
    float fy = (__fmul_rn(Y, Yn) > 0.008856f)
        ? cbrtf(__fmul_rn(Y, inv_Yn))
        : __fmaf_rn(7.787f, __fmul_rn(Y, inv_Yn), 16.0f / 116.0f);
    float fz = (__fmul_rn(Z, Zn) > 0.008856f)
        ? cbrtf(__fmul_rn(Z, inv_Zn))
        : __fmaf_rn(7.787f, __fmul_rn(Z, inv_Zn), 16.0f / 116.0f);

    L = 116.0f * fy - 16.0f;
    a = 500.0f * (fx - fy);
    b = 200.0f * (fy - fz);
}

// =============================================================================
// CUDA KERNELS
// =============================================================================

/// @brief Black out the red channel from the video and add EPITA's logo
/// @param buffer Video buffer
/// @param width Image width
/// @param height Image height
/// @param stride Row stride in bytes
__global__ void remove_red_channel_inp(std::byte *buffer, int width, int height,
                                       int stride)
{
    int x = __fmaf_rn(blockIdx.x, blockDim.x, threadIdx.x);
    int y = __fmaf_rn(blockIdx.y, blockDim.y, threadIdx.y);

    if (x >= width || y >= height)
        return;

    rgb *lineptr = (rgb *)(buffer + y * stride);

    if (y < logo_height && x < logo_width)
    {
        float alpha = __fdiv_rn(logo[y * logo_width + x], 255.0f);
        float inv_alpha = 1.0f - alpha;
        lineptr[x].r = 0;
        lineptr[x].g =
            __float2uint_rn(__fmaf_rn(alpha, lineptr[x].g, inv_alpha * 255.0f));
        lineptr[x].b =
            __float2uint_rn(__fmaf_rn(alpha, lineptr[x].b, inv_alpha * 255.0f));
    }
    else
    {
        lineptr[x].r = 0;
    }
}

/// @brief Apply motion mask to original image with red channel enhancement
__global__ void apply_motion_mask_kernel(uint8_t *original_buffer,
                                         const uint8_t *mask_buffer, int width,
                                         int height, int stride,
                                         int pixel_stride)
{
    int x = __fmaf_rn(blockIdx.x, blockDim.x, threadIdx.x);
    int y = __fmaf_rn(blockIdx.y, blockDim.y, threadIdx.y);

    if (x >= width || y >= height)
        return;

    int idx = y * stride + x * pixel_stride;

    // Check if this pixel has motion (mask is non-zero)
    uint8_t mask_val = mask_buffer[idx];

    if (mask_val > 0) // Motion detected
    {
        original_buffer[idx] = (original_buffer[idx] + 255) >> 1; // R
        original_buffer[idx + 1] = original_buffer[idx + 1] >> 1; // G
        original_buffer[idx + 2] = original_buffer[idx + 2] >> 1; // B
    }
    // If no motion, leave original pixel unchanged
}

/// @brief Background estimation kernel using LAB color space
__global__ void background_estimation_kernel(uint8_t *rgb, PixelState *states,
                                             int width, int height, int stride,
                                             int pixel_stride)
{
    int x = __fmaf_rn(blockIdx.x, blockDim.x, threadIdx.x);
    int y = __fmaf_rn(blockIdx.y, blockDim.y, threadIdx.y);

    if (x >= width || y >= height)
        return;

    // Get current pixel RGB values
    int index = y * stride + x * pixel_stride;
    uint8_t R = rgb[index];
    uint8_t G = rgb[index + 1];
    uint8_t B = rgb[index + 2];

    // Convert to LAB color space
    float L, a, b;
    rgb_to_lab(R, G, B, L, a, b);

    // Get pixel state
    int idx = __fmaf_rn(y, width, x);
    PixelState &s = states[idx];

    if (!s.initialized)
    {
        // Initialize background with first frame
        s.bg_L = L;
        s.bg_a = a;
        s.bg_b = b;
        s.t = 0;
        s.initialized = 1;
    }
    else
    {
        // Calculate distance between background and current frame
        float dL = s.bg_L - L;
        float da = s.bg_a - a;
        float db = s.bg_b - b;
        float dist = __fsqrt_rn(dL * dL + da * da + db * db);
        bool match = dist < 25.0f;

        if (!match)
        {
            // Pixel doesn't match background
            if (s.t == 0)
            {
                // Start new candidate
                s.cand_L = L;
                s.cand_a = a;
                s.cand_b = b;
                s.t++;
            }
            else if (s.t < 50)
            {
                // Update candidate with running average
                s.cand_L = __fmaf_rn(s.cand_L, 0.5f, L * 0.5f);
                s.cand_a = __fmaf_rn(s.cand_a, 0.5f, a * 0.5f);
                s.cand_b = __fmaf_rn(s.cand_b, 0.5f, b * 0.5f);
                s.t++;
            }
            else
            {
                // Accept candidate as new background
                s.bg_L = s.cand_L;
                s.bg_a = s.cand_a;
                s.bg_b = s.cand_b;
                s.t = 0;
            }
        }
        else
        {
            // Pixel matches background - update with weighted average
            s.bg_L = __fmaf_rn(s.bg_L, 0.8f, L * 0.2f);
            s.bg_a = __fmaf_rn(s.bg_a, 0.8f, a * 0.2f);
            s.bg_b = __fmaf_rn(s.bg_b, 0.8f, b * 0.2f);
            s.t = 0;
        }

        // Output distance scaled to [0, 255]
        rgb[index] = __float2uint_rn(fminf(fmaxf(dist * 2.55f, 0.0f), 255.0f));
    }
}

/// @brief Morphological erosion kernel
__global__ void erosion_kernel(uint8_t *input, uint8_t *output, int width,
                               int height, int stride, int pixel_stride)
{
    int x = __fmaf_rn(blockIdx.x, blockDim.x, threadIdx.x);
    int y = __fmaf_rn(blockIdx.y, blockDim.y, threadIdx.y);

    if (x < 1 || x >= width - 1 || y < 1 || y >= height - 1)
        return;

    uint8_t min_val = 255;

    // 3x3 neighborhood
#pragma unroll
    for (int dy = -1; dy <= 1; ++dy)
    {
#pragma unroll
        for (int dx = -1; dx <= 1; ++dx)
        {
            int nx = x + dx;
            int ny = y + dy;
            int idx = ny * stride + nx * pixel_stride;
            min_val = min(min_val, input[idx]);
        }
    }

    // Set all RGB channels to the same value
    int out_idx = y * stride + x * pixel_stride;
    output[out_idx] = min_val;
    output[out_idx + 1] = min_val;
    output[out_idx + 2] = min_val;
}

/// @brief Morphological dilation kernel
__global__ void dilation_kernel(uint8_t *input, uint8_t *output, int width,
                                int height, int stride, int pixel_stride)
{
    int x = __fmaf_rn(blockIdx.x, blockDim.x, threadIdx.x);
    int y = __fmaf_rn(blockIdx.y, blockDim.y, threadIdx.y);

    if (x < 1 || x >= width - 1 || y < 1 || y >= height - 1)
        return;

    uint8_t max_val = 0;

    // 3x3 neighborhood
#pragma unroll
    for (int dy = -1; dy <= 1; ++dy)
    {
#pragma unroll
        for (int dx = -1; dx <= 1; ++dx)
        {
            int nx = x + dx;
            int ny = y + dy;
            int idx = ny * stride + nx * pixel_stride;
            max_val = max(max_val, input[idx]);
        }
    }

    // Set all RGB channels to the same value
    int out_idx = y * stride + x * pixel_stride;
    output[out_idx] = max_val;
    output[out_idx + 1] = max_val;
    output[out_idx + 2] = max_val;
}

/// @brief BFS-based hysteresis initialization
__global__ void bfs_hysteresis_init(const uint8_t *input, uint8_t *output,
                                    Point *queue, int *queue_size, int width,
                                    int height, int stride, int pixel_stride)
{
    int idx = __fmaf_rn(blockIdx.x, blockDim.x, threadIdx.x);
    int total_pixels = width * height;

    if (idx >= total_pixels)
        return;

    int y = idx / width;
    int x = idx % width;
    int pixel_idx = y * stride + x * pixel_stride;

    uint8_t val = input[pixel_idx];

    if (val >= HIGH_THRESHOLD)
    {
        output[pixel_idx] = 255;
        // Add to queue atomically
        int pos = atomicAdd(queue_size, 1);
        if (pos < total_pixels)
        {
            queue[pos] = { x, y };
        }
    }
    else
    {
        output[pixel_idx] = 0;
    }
}

/// @brief BFS-based hysteresis propagation
__global__ void bfs_hysteresis_propagate(const uint8_t *input, uint8_t *output,
                                         const Point *current_queue,
                                         int current_size, Point *next_queue,
                                         int *next_size, int width, int height,
                                         int stride, int pixel_stride)
{
    int idx = __fmaf_rn(blockIdx.x, blockDim.x, threadIdx.x);

    if (idx >= current_size)
        return;

    Point p = current_queue[idx];

// Check 8-connected neighbors
#pragma unroll
    for (int dy = -1; dy <= 1; ++dy)
    {
#pragma unroll
        for (int dx = -1; dx <= 1; ++dx)
        {
            if (dx == 0 && dy == 0)
                continue;

            int nx = p.x + dx;
            int ny = p.y + dy;

            if (nx >= 0 && nx < width && ny >= 0 && ny < height)
            {
                int nidx = ny * stride + nx * pixel_stride;
                uint8_t val = input[nidx];

                if (val >= LOW_THRESHOLD && val < HIGH_THRESHOLD
                    && output[nidx] == 0)
                {
                    // Simple assignment - race condition is acceptable here
                    // as all threads would write the same value (255)
                    output[nidx] = 255;

                    // Add to next queue
                    int pos = atomicAdd(next_size, 1);
                    if (pos < width * height)
                    {
                        next_queue[pos] = { nx, ny };
                    }
                }
            }
        }
    }
}

/// @brief Finalize hysteresis mask by copying to all RGB channels
__global__ void finalize_hysteresis_mask(uint8_t *buffer, const uint8_t *mask,
                                         int width, int height, int stride,
                                         int pixel_stride)
{
    int x = __fmaf_rn(blockIdx.x, blockDim.x, threadIdx.x);
    int y = __fmaf_rn(blockIdx.y, blockDim.y, threadIdx.y);

    if (x >= width || y >= height)
        return;

    int idx = y * stride + x * pixel_stride;
    uint8_t val = mask[idx];
    buffer[idx] = val;
    buffer[idx + 1] = val;
    buffer[idx + 2] = val;
}

// =============================================================================
// CUDA WRAPPER FUNCTIONS
// =============================================================================
void cuda_bfs_hysteresis(uint8_t *buffer, int width, int height, int stride,
                         int pixel_stride)
{
    size_t size = height * stride;
    int total_pixels = width * height;

    // Allocate device memory
    uint8_t *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, buffer, size, hipMemcpyHostToDevice);

    // Initialize BFS queue if not already done
    if (!d_queue)
    {
        hipMalloc(&d_queue, total_pixels * sizeof(Point));
        hipMalloc(&d_next_queue, total_pixels * sizeof(Point));
        hipMalloc(&d_queue_size, sizeof(int));
        hipMalloc(&d_next_queue_size, sizeof(int));
    }

    // Reset queue sizes
    hipMemset(d_queue_size, 0, sizeof(int));
    hipMemset(d_next_queue_size, 0, sizeof(int));

    // Setup grid dimensions
    dim3 block(256);
    dim3 grid((total_pixels + 255) / 256);
    dim3 grid2d((width + 15) / 16, (height + 15) / 16);
    dim3 block2d(16, 16);

    // Initialize with high threshold and populate initial queue
    bfs_hysteresis_init<<<grid, block>>>(d_input, d_output, d_queue,
                                         d_queue_size, width, height, stride,
                                         pixel_stride);

    // BFS propagation
    Point *current_queue = d_queue;
    Point *next_queue = d_next_queue;
    int *current_size = d_queue_size;
    int *next_size = d_next_queue_size;

    for (int iter = 0; iter < 100; ++iter)
    {
        int h_current_size;
        hipMemcpy(&h_current_size, current_size, sizeof(int),
                   hipMemcpyDeviceToHost);

        if (h_current_size == 0)
            break;

        hipMemset(next_size, 0, sizeof(int));

        dim3 prop_grid((h_current_size + 255) / 256);
        bfs_hysteresis_propagate<<<prop_grid, block>>>(
            d_input, d_output, current_queue, h_current_size, next_queue,
            next_size, width, height, stride, pixel_stride);

        // Swap queues
        std::swap(current_queue, next_queue);
        std::swap(current_size, next_size);
    }

    // Finalize and copy back
    finalize_hysteresis_mask<<<grid2d, block2d>>>(d_input, d_output, width,
                                                  height, stride, pixel_stride);
    hipMemcpy(buffer, d_input, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
}

void cuda_opening(uint8_t *buffer, int width, int height, int stride,
                  int pixel_stride)
{
    size_t buffer_size = height * stride;

    // Allocate device memory
    uint8_t *d_input, *d_eroded, *d_opened;
    hipMalloc(&d_input, buffer_size);
    hipMalloc(&d_eroded, buffer_size);
    hipMalloc(&d_opened, buffer_size);

    hipMemcpy(d_input, buffer, buffer_size, hipMemcpyHostToDevice);

    // Setup grid and block dimensions
    dim3 block(16, 16);
    dim3 grid((width + 15) / 16, (height + 15) / 16);

    // Perform opening (erosion followed by dilation)
    erosion_kernel<<<grid, block>>>(d_input, d_eroded, width, height, stride,
                                    pixel_stride);
    dilation_kernel<<<grid, block>>>(d_eroded, d_opened, width, height, stride,
                                     pixel_stride);

    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(buffer, d_opened, buffer_size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_input);
    hipFree(d_eroded);
    hipFree(d_opened);
}

// =============================================================================
// MAIN FILTER IMPLEMENTATION
// =============================================================================

extern "C"
{
    void filter_impl(uint8_t *h_buffer, int width, int height, int plane_stride,
                     int pixel_stride)
    {
        // Create a copy of the original buffer to preserve it
        size_t buffer_size = height * plane_stride;
        uint8_t *h_original_buffer = (uint8_t *)malloc(buffer_size);
        memcpy(h_original_buffer, h_buffer, buffer_size);

        // Initialize pixel states on first call
        if (!d_states)
        {
            image_size = width * height;
            hipMalloc(&d_states, image_size * sizeof(PixelState));
            hipMemset(d_states, 0, image_size * sizeof(PixelState));
        }

        // Allocate device buffer and copy input data
        uint8_t *d_buffer;
        hipMalloc(&d_buffer, height * plane_stride);
        hipMemcpy(d_buffer, h_buffer, height * plane_stride,
                   hipMemcpyHostToDevice);

        // Setup grid and block dimensions
        dim3 block(16, 16);
        dim3 grid((width + 15) / 16, (height + 15) / 16);

        // Apply background estimation
        background_estimation_kernel<<<grid, block>>>(
            d_buffer, d_states, width, height, plane_stride, pixel_stride);

        // Copy intermediate result back to host for morphological operations
        hipMemcpy(h_buffer, d_buffer, height * plane_stride,
                   hipMemcpyDeviceToHost);

        // Apply morphological opening
        cuda_opening(h_buffer, width, height, plane_stride, pixel_stride);

        // Apply hysteresis thresholding
        cuda_bfs_hysteresis(h_buffer, width, height, plane_stride,
                            pixel_stride);

        // Apply the mask to the original image with red channel enhancement
        uint8_t *d_original_buffer, *d_final_mask;
        hipMalloc(&d_original_buffer, buffer_size);
        hipMalloc(&d_final_mask, buffer_size);

        hipMemcpy(d_original_buffer, h_original_buffer, buffer_size,
                   hipMemcpyHostToDevice);
        hipMemcpy(d_final_mask, h_buffer, buffer_size, hipMemcpyHostToDevice);

        // Apply motion mask with red highlighting
        apply_motion_mask_kernel<<<grid, block>>>(d_original_buffer,
                                                  d_final_mask, width, height,
                                                  plane_stride, pixel_stride);
        // Copy the final result back to host
        hipMemcpy(h_buffer, d_original_buffer, buffer_size,
                   hipMemcpyDeviceToHost);
        // Cleanup
        hipFree(d_buffer);
        hipFree(d_original_buffer);
        hipFree(d_final_mask);
        free(h_original_buffer);
    }

} // extern "C"
